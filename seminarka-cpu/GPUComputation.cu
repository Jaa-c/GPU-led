
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

static void HandleError( hipError_t error, const char *file, int line )
{
	if (error != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( error ), file, line );
		//exit( EXIT_FAILURE );
	}
}
#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

